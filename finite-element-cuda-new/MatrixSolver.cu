#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <fstream>
#include <Eigen/Dense>
#include <hipsolver.h>
#include "cusolver_utils.h"



__global__ void assignMemory(double  matrix[], double a[], int rows, int cols)//����������
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= rows || col >= cols) {
        return;
    }
    double temp = matrix[col * rows + row];
    a[col * rows + row] = temp;
}

extern "C" void solveMatrix(Eigen::MatrixXd & kMatrix, Eigen::MatrixXd & fMatrix, Eigen::MatrixXd & uvMatrix) {
    std::ofstream logs("logs.txt", std::ios::app);
    int size = kMatrix.rows();
    const int ldk = size; // ����k��leading dimension
    const int ldf = size; //����F��leading dimension
    double* h_K, * h_F;
    CUDA_CHECK(hipMallocManaged(&h_K, size * size * sizeof(double)));
    CUDA_CHECK(hipMallocManaged(&h_F, size * sizeof(double)));
    //��eigen����ֵ���´���������
    dim3 blockSize(32, 32);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x, (size + blockSize.y - 1) / blockSize.y);
    double* kMatrixData = kMatrix.data();
    double* deviceKMatrixData;
    CUDA_CHECK(hipMallocManaged(&deviceKMatrixData, kMatrix.size() * sizeof(double)));
    CUDA_CHECK(hipMemcpy(deviceKMatrixData, kMatrixData, kMatrix.size() * sizeof(double), hipMemcpyHostToDevice));
    assignMemory << <gridSize, blockSize >> > (deviceKMatrixData, h_K, kMatrix.rows(), kMatrix.cols());
    hipDeviceSynchronize();
    std::ofstream f("h_K.txt");
    if (f.is_open()) {
        for (int i = 0; i < size; i++) {
            for (int j = 0; j < size; j++) {
                double temp = h_K[j * size + i];
                f << temp << "\t";
            }
            f << "\n";
        }
        f.close();
    }
    else {
        logs << "h_K ��ʧ��" << std::endl;
    }
    
    dim3 blockSizeF = dim3(1, 32);
    dim3 gridSizeF =  dim3(1, (size + blockSize.y - 1) / blockSize.y);
    double* fMatrixData = fMatrix.data();
    double* deviceFMatrixData;
    CUDA_CHECK(hipMallocManaged(&deviceFMatrixData, fMatrix.size() * sizeof(double)));
    CUDA_CHECK(hipMemcpy(deviceFMatrixData, fMatrixData, fMatrix.size() * sizeof(double), hipMemcpyHostToDevice));
    assignMemory << <gridSizeF, blockSizeF >> > (deviceFMatrixData, h_F, fMatrix.rows(), fMatrix.cols());
    hipDeviceSynchronize();
    f = std::ofstream("h_F.txt");
    if (f.is_open()) {
        for (int i = 0; i < size; i++) {
            double temp = h_F[i];
            f << temp << "\n";
        }
        f.close();
    }
    else {
        logs << "h_F ��ʧ��" << std::endl;
    }
      
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int m = size;
    const int lda = m;
    const int ldb = m;


    std::vector<double> X(m, 0);
    std::vector<double> LU(lda * m, 0);
    std::vector<int> Ipiv(m, 0);
    int info = 0;

    double* d_A = nullptr; /* device copy of A */
    double* d_B = nullptr; /* device copy of B */
    int* d_Ipiv = nullptr; /* pivoting sequence */
    int* d_info = nullptr; /* error info */

    int lwork = 0;            /* size of workspace */
    double* d_work = nullptr; /* device workspace for getrf */

    const int pivot_on = 0;

    if (pivot_on) {
        printf("pivot is on : compute P*A = L*U \n");
    }
    else {
        printf("pivot is off: compute A = L*U (not numerically stable)\n");
    }


    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(double) * size * size));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), sizeof(double) * size));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_Ipiv), sizeof(int) * Ipiv.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, h_K, sizeof(double) * size * size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(
        hipMemcpyAsync(d_B, h_F, sizeof(double) *  size, hipMemcpyHostToDevice, stream));

    /* step 3: query working space of getrf */
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_work), sizeof(double) * lwork));

    /* step 4: LU factorization */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info));
    }
    else {
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, m, m, d_A, lda, d_work, NULL, d_info));
    }

    if (pivot_on) {
        CUDA_CHECK(hipMemcpyAsync(Ipiv.data(), d_Ipiv, sizeof(int) * Ipiv.size(),
            hipMemcpyDeviceToHost, stream));
    }
    CUDA_CHECK(
        hipMemcpyAsync(LU.data(), d_A, sizeof(double) * size * size, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    if (pivot_on) {
        printf("pivoting sequence, matlab base-1\n");
        for (int j = 0; j < m; j++) {
            printf("Ipiv(%d) = %d\n", j + 1, Ipiv[j]);
        }
    }



    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
            d_A, lda, d_Ipiv, d_B, ldb, d_info));
    }
    else {
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, /* nrhs */
            d_A, lda, NULL, d_B, ldb, d_info));
    }

    CUDA_CHECK(
        hipMemcpyAsync(X.data(), d_B, sizeof(double) * X.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    uvMatrix = Eigen::MatrixXd::Zero(size, 1);
    f = std::ofstream("h_UV.txt");
    if (f.is_open()) {
        for (int i = 0; i < size; i++) {
            double temp = X[i];
            uvMatrix(i, 0) = temp;
            f << temp << "\n";
        }
        f.close();
    }

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_Ipiv));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());
    logs.close();

}


